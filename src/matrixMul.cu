#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstring>
#include <stdlib.h>


const int THREADS = 16;
const int MOD = 1000;

/**
 ����˷�
*/
__global__ void mulMatrixKernel(int *c, const int *a, const int *b, int size, size_t pitch)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = i / size;
	const int column = i % size;
	if (row < size && column < size) {
		int tmp = 0;
		// Ӧ��ʹ�ö������г�����Ϊ��λ
		for (int i = 0; i < size; ++i)
			tmp += a[row * pitch + i] * b[i * pitch + column];
		c[row * pitch + column] = tmp;
	}
}

__host__ void mulMatrix2D(int *c, const int *a, const int *b, int size)
{
	int *matrix_a;
	int *matrix_b;
	int *matrix_c;
	size_t pitch;
	// ��֤������ڴ��Ǻ������ģ����������ϵ��ڴ���ʣ���˿��Ա�֤���з���ʱ�������ŵ�Ч��
	hipMallocPitch((void **)&matrix_a, &pitch, sizeof(int) * size, size);
	hipMallocPitch((void **)&matrix_b, &pitch, sizeof(int) * size, size);
	hipMallocPitch((void **)&matrix_c, &pitch, sizeof(int) * size, size);

	hipMemcpy2D(matrix_a, sizeof(int) * size, a, pitch, sizeof(int) * size, size, hipMemcpyHostToDevice);
	hipMemcpy2D(matrix_b, sizeof(int) * size, b, pitch, sizeof(int) * size, size, hipMemcpyHostToDevice);

	int threadsPerBlock = THREADS;
	int blocksPerGrid = (size * size + THREADS - 1) / THREADS;

	mulMatrixKernel <<<blocksPerGrid, threadsPerBlock>>> (matrix_c, matrix_a, matrix_b, size, pitch);

	hipMemcpy2D(c, sizeof(int) * size, matrix_c, pitch, sizeof(int) * size, size, hipMemcpyDeviceToHost);
	
	// cpu���ڴ���δ�������
	for (int i = 0; i < size; ++i) {
		for (int j = 0; j < size; ++j)
			printf("%d ", c[i * size + j]);
		printf("\n");
	}

	hipFree(matrix_a);
	hipFree(matrix_b);
	hipFree(matrix_c);
}

/**
 ���ξ����������
*/
__host__ void integerMatrixGenerate(int *a, int size)
{
	a = (int *)malloc(sizeof(int) * size * size);
	for (int i = 0; i < size; i++) 
		for (int j = 0; j < size; j++) 
			a[i * size + j] = rand() % MOD;
}

int main()
{
	int *a, *b, *c;
	const int size = 4;
	integerMatrixGenerate(a, size);
	integerMatrixGenerate(b, size);
	mulMatrix2D(c, a, b, size);

    return 0;
}



/**
 ģ���ṩ�������ӷ�����
*/
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}


__host__ void testAddVector()
{
	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return;
	}

	printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return;
	}

	return;
}

